
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
#include <stdint.h>

__device__ hipFloatComplex detect(const hipFloatComplex a) {
    return make_hipFloatComplex((a.x * a.x) + (a.y * a.y), 0.0f);
}

template<uint64_t NBEAMS, uint64_t NANTS, uint64_t NCHANS,
         uint64_t NTIME, uint64_t NPOLS, uint64_t TBLOCK,
         bool EnableIncoherentBeam, bool EnableIncoherentBeamSqrt>
__global__ void ATA(const hipFloatComplex* input,
                    const hipFloatComplex* phasor,
                          hipFloatComplex* out) {
    int bi = threadIdx.x;
    int ti = bi + (blockIdx.y * TBLOCK);
    int ch = blockIdx.x;

    // Load the phasors to shared memory.
    __shared__ hipFloatComplex phr_cache[NBEAMS][NANTS][NPOLS];

    int iy = (ch * NPOLS) + (bi * NPOLS * NCHANS * NANTS);
    const int dy = NPOLS * NCHANS;

    if (bi < NBEAMS) {
        for (int a = 0; a < NANTS; a++, iy += dy) {
            phr_cache[bi][a][0] = phasor[iy+0];
            phr_cache[bi][a][1] = phasor[iy+1];
        }
    }

    __syncthreads();

    // Load the antenna values to registers.
    hipFloatComplex ant_cache[NANTS][NPOLS];

    int ix = (ch * NTIME * NPOLS) + (ti * NPOLS);
    const int dx = NTIME * NCHANS * NPOLS;

    for (int a = 0; a < NANTS; a++, ix += dx) {
        ant_cache[a][0] = input[ix+0];
        ant_cache[a][1] = input[ix+1];
    }

    // Multiply and accumulate.
    int iz = (ch * NTIME) + ti;
    const int dz = NTIME * NCHANS;

    for (int b = 0; b < NBEAMS; b++, iz += dz) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0; a < NANTS; a++) {
            acc[0] = hipCaddf(acc[0], hipCmulf(ant_cache[a][0], phr_cache[b][a][0]));
            acc[1] = hipCaddf(acc[1], hipCmulf(ant_cache[a][1], phr_cache[b][a][1]));
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }

    if (EnableIncoherentBeam) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0; a < NANTS; a++) {
            acc[0] = hipCaddf(acc[0], detect(hipCmulf(ant_cache[a][0], phr_cache[0][a][0])));
            acc[1] = hipCaddf(acc[1], detect(hipCmulf(ant_cache[a][1], phr_cache[0][a][1])));
        }

        if (EnableIncoherentBeamSqrt) {
            acc[0] = make_hipFloatComplex(sqrt(acc[0].x), acc[0].y);
            acc[1] = make_hipFloatComplex(sqrt(acc[1].x), acc[1].y);
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }
}

template<uint64_t NBEAMS, uint64_t NANTS, uint64_t NCHANS,
         uint64_t NTIME, uint64_t NPOLS, uint64_t TBLOCK,
         bool EnableIncoherentBeam, bool EnableIncoherentBeamSqrt>
__global__ void MeerKAT(const hipFloatComplex* input,
                        const hipFloatComplex* phasor,
                              hipFloatComplex* out) {
    int bi = threadIdx.x;
    int ti = bi + (blockIdx.y * TBLOCK);
    int ch = blockIdx.x;

    // Load the antenna values to registers.
    hipFloatComplex ant_cache[NANTS][NPOLS];

    int ix = (ch * NTIME * NPOLS) + (ti * NPOLS);
    const int dx = NTIME * NCHANS * NPOLS;

    for (int a = 0; a < NANTS; a++, ix += dx) {
        ant_cache[a][0] = input[ix+0];
        ant_cache[a][1] = input[ix+1];
    }

    // Multiply and accumulate.
    int iy = 0;
    int iz = (ch * NTIME) + ti;
    const int dz = NTIME * NCHANS;

    for (int b = 0; b < NBEAMS; b++, iz += dz) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0, x = ix; a < NANTS; a++, iy += 1, x += dx) {
            acc[0] = hipCaddf(acc[0], hipCmulf(ant_cache[a][0], phasor[iy]));
            acc[1] = hipCaddf(acc[1], hipCmulf(ant_cache[a][1], phasor[iy]));
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }
}
