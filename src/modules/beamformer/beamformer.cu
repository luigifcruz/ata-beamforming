#include "hip/hip_runtime.h"
#include "hip/hip_complex.h"

// TODO: Convert to Ops.

#include "blade/memory/base.hh"

using namespace Blade;

__device__ hipFloatComplex detect(const hipFloatComplex a) {
    return make_hipFloatComplex((a.x * a.x) + (a.y * a.y), 0.0f);
}

template<U64 NBEAMS, U64 NANTS, U64 NCHANS,
         U64 NTIME,  U64 NPOLS, U64 TBLOCK,
         BOOL EnableIncoherentBeam, BOOL EnableIncoherentBeamSqrt>
__global__ void ATA(const hipFloatComplex* input,
                    const hipFloatComplex* phasor,
                          hipFloatComplex* out) {
    int bi = threadIdx.x;
    int ti = bi + (blockIdx.y * TBLOCK);
    int ch = blockIdx.x;

    // Load the phasors to shared memory.
    __shared__ hipFloatComplex phr_cache[NBEAMS][NANTS][NPOLS];

    int iy = (ch * NPOLS) + (bi * NPOLS * NCHANS * NANTS);
    const int dy = NPOLS * NCHANS;

    if (bi < NBEAMS) {
        for (int a = 0; a < NANTS; a++, iy += dy) {
            phr_cache[bi][a][0] = phasor[iy+0];
            phr_cache[bi][a][1] = phasor[iy+1];
        }
    }

    __syncthreads();

    // Load the antenna values to registers.
    hipFloatComplex ant_cache[NANTS][NPOLS];

    int ix = (ch * NTIME * NPOLS) + (ti * NPOLS);
    const int dx = NTIME * NCHANS * NPOLS;

    for (int a = 0; a < NANTS; a++, ix += dx) {
        ant_cache[a][0] = input[ix+0];
        ant_cache[a][1] = input[ix+1];
    }

    // Multiply and accumulate.
    int iz = (ch * NTIME) + ti;
    const int dz = NTIME * NCHANS;

    for (int b = 0; b < NBEAMS; b++, iz += dz) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0; a < NANTS; a++) {
            acc[0] = hipCaddf(acc[0], hipCmulf(ant_cache[a][0], phr_cache[b][a][0]));
            acc[1] = hipCaddf(acc[1], hipCmulf(ant_cache[a][1], phr_cache[b][a][1]));
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }

    if (EnableIncoherentBeam) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0; a < NANTS; a++) {
            acc[0] = hipCaddf(acc[0], detect(hipCmulf(ant_cache[a][0], phr_cache[0][a][0])));
            acc[1] = hipCaddf(acc[1], detect(hipCmulf(ant_cache[a][1], phr_cache[0][a][1])));
        }

        if (EnableIncoherentBeamSqrt) {
            acc[0] = make_hipFloatComplex(sqrt(acc[0].x), acc[0].y);
            acc[1] = make_hipFloatComplex(sqrt(acc[1].x), acc[1].y);
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }
}

template<U64 NBEAMS, U64 NANTS, U64 NCHANS,
         U64 NTIME, U64 NPOLS, U64 TBLOCK,
         BOOL EnableIncoherentBeam, BOOL EnableIncoherentBeamSqrt>
__global__ void MeerKAT(const hipFloatComplex* input,
                        const hipFloatComplex* phasor,
                              hipFloatComplex* out) {
    int bi = threadIdx.x;
    int ti = bi + (blockIdx.y * TBLOCK);
    int ch = blockIdx.x;

    // Load the antenna values to registers.
    hipFloatComplex ant_cache[NANTS][NPOLS];

    int ix = (ch * NTIME * NPOLS) + (ti * NPOLS);
    const int dx = NTIME * NCHANS * NPOLS;

    for (int a = 0; a < NANTS; a++, ix += dx) {
        ant_cache[a][0] = input[ix+0];
        ant_cache[a][1] = input[ix+1];
    }

    // Multiply and accumulate.
    int iy = 0;
    int iz = (ch * NTIME) + ti;
    const int dz = NTIME * NCHANS;

    for (int b = 0; b < NBEAMS; b++, iz += dz) {
        hipFloatComplex acc[NPOLS] = {{0.0, 0.0}};

        for (int a = 0, x = ix; a < NANTS; a++, iy += 1, x += dx) {
            acc[0] = hipCaddf(acc[0], hipCmulf(ant_cache[a][0], phasor[iy]));
            acc[1] = hipCaddf(acc[1], hipCmulf(ant_cache[a][1], phasor[iy]));
        }

        reinterpret_cast<float4*>(out)[iz] = *reinterpret_cast<float4*>(acc);
    }
}
