
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>
#include "hip/hip_complex.h"

template<uint64_t N, uint64_t NCHAN, uint64_t INTG>
__global__ void detector_4pol(const hipFloatComplex* input, float* output) {
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < (N / INTG)) {
        reinterpret_cast<float4*>(output)[tid] = {0.0, 0.0, 0.0, 0.0};
    }

    __syncthreads();

    if (tid < N) {
        const float4 sample = reinterpret_cast<const float4*>(input)[tid];
        
        hipFloatComplex sample_X = make_hipFloatComplex(sample.x, sample.y);
        hipFloatComplex sample_Y = make_hipFloatComplex(sample.z, sample.w);

        hipFloatComplex X = hipCmulf(sample_X, hipConjf(sample_X));
        hipFloatComplex Y = hipCmulf(sample_Y, hipConjf(sample_Y));
        hipFloatComplex Z = hipCmulf(sample_X, hipConjf(sample_Y));

        const uint64_t oid = (tid / INTG) * 4;
        atomicAdd(output + oid + 0, X.x);
        atomicAdd(output + oid + 1, Y.x);
        atomicAdd(output + oid + 2, Z.x);
        atomicAdd(output + oid + 3, Z.y);
    }
}

template<uint64_t N, uint64_t NCHAN, uint64_t INTG>
__global__ void detector_1pol(const hipFloatComplex* input, float* output) {
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < (N / INTG)) {
        output[tid] = 0.0;
    }

    __syncthreads();

    if (tid < N) {
        const float4 sample = reinterpret_cast<const float4*>(input)[tid];
        
        hipFloatComplex sample_X = make_hipFloatComplex(sample.x, sample.y);
        hipFloatComplex sample_Y = make_hipFloatComplex(sample.z, sample.w);

        hipFloatComplex X = hipCmulf(sample_X, hipConjf(sample_X));
        hipFloatComplex Y = hipCmulf(sample_Y, hipConjf(sample_Y));

        atomicAdd(output + (tid / INTG), X.x + Y.x);
    }
}
