
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdint.h>
#include "hip/hip_complex.h"

// TODO: Convert to Ops.

template<uint64_t N, uint64_t INTG>
__global__ void detector_4pol(const hipFloatComplex* input,
                              float* output,
                              const bool* resetTensor) {
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (*resetTensor) {
        if (tid < (N / INTG)) {
            reinterpret_cast<float4*>(output)[tid] = {0.0, 0.0, 0.0, 0.0};
        }
        __syncthreads();
    }

    if (tid < N) {
        const float4 sample = reinterpret_cast<const float4*>(input)[tid];
        
        const hipFloatComplex sample_X = make_hipFloatComplex(sample.x, sample.y);
        const hipFloatComplex sample_Y = make_hipFloatComplex(sample.z, sample.w);

        const hipFloatComplex X = hipCmulf(sample_X, hipConjf(sample_X));
        const hipFloatComplex Y = hipCmulf(sample_Y, hipConjf(sample_Y));
        const hipFloatComplex Z = hipCmulf(sample_X, hipConjf(sample_Y));

        const uint64_t oid = (tid / INTG) * 4;
        atomicAdd(output + oid + 0, X.x);
        atomicAdd(output + oid + 1, Y.x);
        atomicAdd(output + oid + 2, Z.x);
        atomicAdd(output + oid + 3, Z.y);
    }
}

template<uint64_t N, uint64_t INTG>
__global__ void detector_1pol(const hipFloatComplex* input,
                              float* output,
                              const bool* resetTensor) {
    const uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (*resetTensor) {
        if (tid < (N / INTG)) {
            output[tid] = 0.0;
        }
        __syncthreads();
    }

    if (tid < N) {
        const float4 sample = reinterpret_cast<const float4*>(input)[tid];

        const float X = sample.x * sample.x + sample.y * sample.y;
        const float Y = sample.z * sample.z + sample.w * sample.w;

        atomicAdd(output + (tid / INTG), X + Y);
    }
}
