
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

template<uint64_t N>
__global__ void shuffle(const hipFloatComplex* input, const uint64_t* indices, hipFloatComplex* output) {
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < N){
        output[tid] = input[indices[tid]];
    }
}
