#include "hip/hip_runtime.h"
#include "blade/memory/base.hh"

//#define DEBUG

using namespace Blade;

// Input Shape:       [A, F, T, P]
// Blocks per Grid:   [A, F / BLOCK_SIZE]
// Threads per Block: [BLOCK_SIZE]
// 
// EXAMPLE (BLOCK_SIZE = 4):
// Input Shape:       [20, 200, 4, 2]
// Blocks per Grid:   [20, 4]
// Threads Per Block: [50]

// 
// Fast but limited by the shared memory size.
//

template<typename IT, typename OT, U64 A, U64 C, U64 T, U64 P, U64 BLOCK_SIZE>
__global__ void correlator_sm(const ArrayTensor<Device::CUDA, IT> input, 
                                    ArrayTensor<Device::CUDA, OT> output) {
    // 1. Load antenna data chunk into shared memory.
    // 2. Do the multiply conjugate (XX = AX * CONJ(BX)).
    // 3. Store the result in the output tensor.

    // Get Block index.
    
    const U64 BIX = blockIdx.x;  // Block Index X
    const U64 BIY = blockIdx.y;  // Block Index Y

    // Get Thread index.

    const U64 TIX = threadIdx.x;  // Thread Index X

    // Calculate constants.

    const U64 OUTPUT_POLS = 4;                // XX, XY, YX, YY
    const U64 AAI = BIX;                      // Antenna A Index
    const U64 CI = TIX + (BIY * BLOCK_SIZE);  // Channel Index 

    // Declare shared memory within block.

    __shared__ IT reference_antenna_data[C][T][P];

    // Load block of input data into shared memory.

    for (U64 TI = 0; TI < T; TI++) {
        for (U64 PI = 0; PI < P; PI++) {
            const U64 ANTENNA_A_INDEX = (AAI * C * T * P) + (CI * T * P) + (TI * P) + PI;
            reference_antenna_data[CI][TI][PI] = input[ANTENNA_A_INDEX];

#ifdef DEBUG
            printf("++ BIX: %ld/%d, BIY: %ld/%d, TIX: %ld || AAI: %ld, CI: %ld, TI: %ld, PI: %ld || ANTENNA_A_INDEX: %ld\n", 
                   BIX, gridDim.x, BIY, gridDim.y, TIX, AAI, CI, TI, PI, ANTENNA_A_INDEX);
#endif
        }
    }
    __syncthreads();

    // Run the correlation and store the result in the output tensor.

    for (U64 ABI = AAI; ABI < A; ABI++) {
        const U64 BASELINE_INDEX = ((AAI * (2 * A - AAI + 1)) / 2) + (ABI - AAI);

        for (U64 TI = 0; TI < T; TI++) {
            const U64 ANTENNA_B_INDEX = (ABI * C * T * P) + (CI * T * P) + (TI * P);

            const IT& AVAX = reference_antenna_data[CI][TI][0];  // Antenna Voltage A Pol X
            const IT& AVAY = reference_antenna_data[CI][TI][1];  // Antenna Voltage A Pol Y

            const IT& AVBX = input[ANTENNA_B_INDEX + 0];  // Antenna Voltage B Pol X
            const IT& AVBY = input[ANTENNA_B_INDEX + 1];  // Antenna Voltage B Pol Y

            const OT XX = AVAX * AVBX.conj();  // XX
            const OT XY = AVAX * AVBY.conj();  // XY
            const OT YX = AVAY * AVBX.conj();  // YX
            const OT YY = AVAY * AVBY.conj();  // YY

            const U64 OUTPUT_INDEX = (BASELINE_INDEX * C * T * OUTPUT_POLS) + (CI * T * OUTPUT_POLS) + (TI * OUTPUT_POLS);

            output[OUTPUT_INDEX + 0] += XX;
            output[OUTPUT_INDEX + 1] += XY;
            output[OUTPUT_INDEX + 2] += YX;
            output[OUTPUT_INDEX + 3] += YY;
                
#ifdef DEBUG
            printf("-- BIX: %ld/%d, BIY: %ld/%d, TIX: %ld || ABI: %ld, CI: %ld, TI: %ld || BASELINE_INDEX: %ld, OUTPUT_INDEX: %ld\n",
                    BIX, gridDim.x, BIY, gridDim.y, TIX, ABI, CI, TI, BASELINE_INDEX, OUTPUT_INDEX);
#endif
        }
    }
}

//
// Global memory version without shared memory.
//

template<typename IT, typename OT, U64 A, U64 C, U64 T, U64 P, U64 BLOCK_SIZE>
__global__ void correlator(const ArrayTensor<Device::CUDA, IT> input, 
                                 ArrayTensor<Device::CUDA, OT> output) {
    // 1. Load antenna A and B data.
    // 2. Do the multiply conjugate (XX = AX * CONJ(BX)).
    // 3. Store the result in the output tensor.

    // Get Block index.
    
    const U64 BIX = blockIdx.x;  // Block Index X
    const U64 BIY = blockIdx.y;  // Block Index Y

    // Get Thread index.

    const U64 TIX = threadIdx.x;  // Thread Index X

    // Calculate constants.

    const U64 OUTPUT_POLS = 4;                // XX, XY, YX, YY
    const U64 AAI = BIX;                      // Antenna A Index
    const U64 CI = TIX + (BIY * BLOCK_SIZE);  // Channel Index

    // Run the correlation and store the result in the output tensor.

    for (U64 ABI = AAI; ABI < A; ABI++) {
        const U64 BASELINE_INDEX = ((AAI * (2 * A - AAI + 1)) / 2) + (ABI - AAI);

        for (U64 TI = 0; TI < T; TI++) {
            const U64 ANTENNA_A_INDEX = (AAI * C * T * P) + (CI * T * P) + (TI * P);

            const IT& AVAX = input[ANTENNA_A_INDEX + 0];  // Antenna Voltage A Pol X
            const IT& AVAY = input[ANTENNA_A_INDEX + 1];  // Antenna Voltage A Pol Y

            const U64 ANTENNA_B_INDEX = (ABI * C * T * P) + (CI * T * P) + (TI * P);

            const IT& AVBX = input[ANTENNA_B_INDEX + 0];  // Antenna Voltage B Pol X
            const IT& AVBY = input[ANTENNA_B_INDEX + 1];  // Antenna Voltage B Pol Y

            const OT XX = AVAX * AVBX.conj();  // XX
            const OT XY = AVAX * AVBY.conj();  // XY
            const OT YX = AVAY * AVBX.conj();  // YX
            const OT YY = AVAY * AVBY.conj();  // YY

            const U64 OUTPUT_INDEX = (BASELINE_INDEX * C * T * OUTPUT_POLS) + (CI * T * OUTPUT_POLS) + (TI * OUTPUT_POLS);

            output[OUTPUT_INDEX + 0] += XX;
            output[OUTPUT_INDEX + 1] += XY;
            output[OUTPUT_INDEX + 2] += YX;
            output[OUTPUT_INDEX + 3] += YY;
                
#ifdef DEBUG
            printf("-- BIX: %ld/%d, BIY: %ld/%d, TIX: %ld || ABI: %ld, CI: %ld, TI: %ld || AAI: %ld, ABI: %ld || BASELINE_INDEX: %ld, OUTPUT_INDEX: %ld\n",
                   BIX, gridDim.x, BIY, gridDim.y, TIX, ABI, CI, TI, ANTENNA_A_INDEX, ANTENNA_B_INDEX, BASELINE_INDEX, OUTPUT_INDEX);
#endif
        }
    }
}

//
// Global memory version with integration.
//

template<typename IT, typename OT, U64 A, U64 C, U64 T, U64 P, U64 BLOCK_SIZE>
__global__ void correlator_integrator(const ArrayTensor<Device::CUDA, IT> input, 
                                            ArrayTensor<Device::CUDA, OT> output) {
    // 1. Load antenna A and B data.
    // 2. Create temporary variables to accumulate the result.
    // 3. Add the multiply conjugate (XX = AX * CONJ(BX)) result to the temporary variables.
    // 4. Store the result in the output tensor.

    // Get Block index.
    
    const U64 BIX = blockIdx.x;  // Block Index X
    const U64 BIY = blockIdx.y;  // Block Index Y

    // Get Thread index.

    const U64 TIX = threadIdx.x;  // Thread Index X

    // Calculate constants.

    const U64 OUTPUT_POLS = 4;                // XX, XY, YX, YY
    const U64 AAI = BIX;                      // Antenna A Index
    const U64 CI = TIX + (BIY * BLOCK_SIZE);  // Channel Index

    // Run the correlation and store the result in the output tensor.

    for (U64 ABI = AAI; ABI < A; ABI++) {
        const U64 BASELINE_INDEX = ((AAI * (2 * A - AAI + 1)) / 2) + (ABI - AAI);
        
        OT sumXX = OT(0.0f, 0.0f);
        OT sumXY = OT(0.0f, 0.0f);
        OT sumYX = OT(0.0f, 0.0f);
        OT sumYY = OT(0.0f, 0.0f);

        for (U64 TI = 0; TI < T; TI++) {
            const U64 ANTENNA_A_INDEX = (AAI * C * T * P) + (CI * T * P) + (TI * P);

            const IT& AVAX = input[ANTENNA_A_INDEX + 0];  // Antenna Voltage A Pol X
            const IT& AVAY = input[ANTENNA_A_INDEX + 1];  // Antenna Voltage A Pol Y

            const U64 ANTENNA_B_INDEX = (ABI * C * T * P) + (CI * T * P) + (TI * P);

            const IT& AVBX = input[ANTENNA_B_INDEX + 0];  // Antenna Voltage B Pol X
            const IT& AVBY = input[ANTENNA_B_INDEX + 1];  // Antenna Voltage B Pol Y

            sumXX += AVAX * AVBX.conj();  // XX
            sumXY += AVAX * AVBY.conj();  // XY
            sumYX += AVAY * AVBX.conj();  // YX
            sumYY += AVAY * AVBY.conj();  // YY
        }

        const U64 OUTPUT_INDEX = (BASELINE_INDEX * C * OUTPUT_POLS) + (CI * OUTPUT_POLS);

        output[OUTPUT_INDEX + 0] += sumXX;
        output[OUTPUT_INDEX + 1] += sumXY;
        output[OUTPUT_INDEX + 2] += sumYX;
        output[OUTPUT_INDEX + 3] += sumYY;

#ifdef DEBUG
        printf("-- BIX: %ld/%d, BIY: %ld/%d, TIX: %ld || ABI: %ld, CI: %ld || AAI: %ld, ABI: %ld || BASELINE_INDEX: %ld, OUTPUT_INDEX: %ld\n",
               BIX, gridDim.x, BIY, gridDim.y, TIX, ABI, CI, AAI, ABI, BASELINE_INDEX, OUTPUT_INDEX);
#endif
    }
}