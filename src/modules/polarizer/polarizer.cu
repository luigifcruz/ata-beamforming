
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hip/hip_fp16.h>
#include <stdint.h>

template<typename IT, typename OT, uint64_t N>
__global__ void polarizer(const IT* input, OT* output) {
    const int tid = (blockIdx.x * blockDim.x + threadIdx.x) * 2;

    if (tid < (N * 2)) {
        // The complex multiplication below can be simplified because
        // the real part of the phasor is 0.0. Boring implementation:
        // const IT yPol90 = cuCmulf(yPol, make_cuFloatComplex(0.0, 1.0));

        const OT& yPol = input[tid + 1];
        const float x = -hipCimagf(yPol);
        const float y = +hipCrealf(yPol);
        const IT yPol90 = make_hipFloatComplex(x, y);

        const IT xPol = input[tid + 0];
        output[tid + 0] = hipCaddf(xPol, yPol90);
        output[tid + 1] = hipCsubf(xPol, yPol90);
    }
}
