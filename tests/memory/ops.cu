#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>

#include <gtest/gtest.h>
#include <hip/hip_complex.h>
#include <hip/hip_fp16.h>

#define BL_OPS_HOST_SIDE_KEY
#include "blade/memory/ops.hh"

using namespace Blade;

//
// Ops Alignment Test
//

TEST(ComplexSizeTest, Half) {
    EXPECT_EQ(sizeof(half2), sizeof(ops::complex<F16>));
}

TEST(ComplexSizeTest, Float) {
    EXPECT_EQ(sizeof(hipFloatComplex), sizeof(ops::complex<F32>));
}

TEST(ComplexSizeTest, Double) {
    EXPECT_EQ(sizeof(hipDoubleComplex), sizeof(ops::complex<F64>));
}

// 
// Ops Test
//

#define EPSILON 1e-2f

__device__ bool assert_eq(half a, float b) {
    if (fabsf(__half2float(a) - b) > EPSILON) {
        printf("Assertion failed: %f != %f\n", __half2float(a), b);
        return true;
    }
    return false;
}

__device__ __managed__ bool err;

__global__ void kadd() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
    ops::complex<F16> c = a + b;

    hipComplex cu_a = make_hipComplex(1.0, 2.0);
    hipComplex cu_b = make_hipComplex(3.0, 4.0);
    hipComplex cu_c = hipCaddf(cu_a, cu_b);

    err |= assert_eq(c.real(), cu_c.x);
    err |= assert_eq(c.imag(), cu_c.y);
}

TEST(HalfComplexTest, OpAdd) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    kadd<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

__global__ void ksub() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
    ops::complex<F16> c = a - b;

    hipComplex cu_a = make_hipComplex(1.0, 2.0);
    hipComplex cu_b = make_hipComplex(3.0, 4.0);
    hipComplex cu_c = hipCsubf(cu_a, cu_b);

    err |= assert_eq(c.real(), cu_c.x);
    err |= assert_eq(c.imag(), cu_c.y);
}

TEST(HalfComplexTest, OpSub) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    ksub<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

__global__ void kmul() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
    ops::complex<F16> c = a * b;

    hipComplex cu_a = make_hipComplex(1.0, 2.0);
    hipComplex cu_b = make_hipComplex(3.0, 4.0);
    hipComplex cu_c = hipCmulf(cu_a, cu_b);

    err |= assert_eq(c.real(), cu_c.x);
    err |= assert_eq(c.imag(), cu_c.y);
}

TEST(HalfComplexTest, OpMul) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    kmul<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

__global__ void kdiv() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
    ops::complex<F16> c = a / b;

    hipComplex cu_a = make_hipComplex(1.0, 2.0);
    hipComplex cu_b = make_hipComplex(3.0, 4.0);
    hipComplex cu_c = hipCdivf(cu_a, cu_b);

    err |= assert_eq(c.real(), cu_c.x);
    err |= assert_eq(c.imag(), cu_c.y);
}

TEST(HalfComplexTest, OpDiv) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    kdiv<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

__global__ void keq() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
 
    err |=  (a == b);
    err |= !(a == a);
}

TEST(HalfComplexTest, OpEq) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    keq<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

__global__ void kieq() {
    ops::complex<F16> a(1.0, 2.0);
    ops::complex<F16> b(3.0, 4.0);
 
    err |= !(a != b);
    err |=  (a != a);
}

TEST(HalfComplexTest, OpIeq) {
    ::testing::GTEST_FLAG(print_time) = true;
    err = false;
    kieq<<<1, 1>>>();
    hipDeviceSynchronize();
    EXPECT_FALSE(err);
}

//
// Test Logic
//

int main(int argc, char** argv) {
	testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
